#include "hip/hip_runtime.h"
﻿#include "./kernel.h"
#include <host_defines.h>




__device__ const int WIDTH = 8192; 
__device__ const int HEIGHT = 4608;
__device__ const int MAX_Iteration = 256;// for color


static __global__ void
//__launch_bounds__(256,10)
kernel(const float lower_left_real, const float lower_left_imag, float factorX, float factorY, uchar4* __restrict__ dp_ptr)
{

    unsigned int x_dim = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y_dim = blockIdx.y * blockDim.y + threadIdx.y;

    //int index = WIDTH * y_dim + x_dim;//;
    int index = ((gridDim.x * blockDim.x))* y_dim + x_dim;


    // 1 / WIDTH = 0.00012207031f
    float c_real = (static_cast<float>(x_dim)* 0.00012207031f) * factorX + lower_left_real;
    float c_imag = (static_cast<float>(y_dim)* 0.00012207031f) * factorY + lower_left_imag;

    float z_real = 0.0f;
    float z_imag = 0.0f;
    float z_real_square = 0.0f;
    float z_imag_square = 0.0f;
    int iteration = 0;
    
 /*
#pragma unroll
    do
    {
        z_real_square = z_real*z_real; //use __fmul_rd more slowly
        z_imag_square = z_imag * z_imag;
        //float real_temp = z_real_square - z_imag_square + c_real; // a^2 - b^2 + c_real
        z_imag = 2.0 * z_real * z_imag + c_imag;                 // 2ab+c_imag
        z_real = z_real_square - z_imag_square + c_real; // a^2 - b^2 + c_real;
        ++iteration;
    } while ((z_real_square+z_imag_square) <= 4.0f && (iteration < MAX_Iteration));
 */    
 
 #pragma unroll
    while ((z_real_square + z_imag_square) <= 4.0f && (iteration < MAX_Iteration))
    {
        z_real_square = z_real * z_real; //use __fmul_rd more slowly
        z_imag_square = z_imag * z_imag;
        //float real_temp = z_real_square - z_imag_square + c_real; // a^2 - b^2 + c_real
        z_imag = 2.0 * z_real * z_imag + c_imag;                 // 2ab+c_imag
        z_real = z_real_square - z_imag_square + c_real; // a^2 - b^2 + c_real;
        ++iteration;
    }



    
    /*
    while (((z_real*z_real) + (z_imag*z_imag)) <= 4.0f && (iteration < MAX_Iteration))
    {
        //Z^2+c
        float real_temp =z_real*z_real - z_imag*z_imag + c_real; // a^2 - b^2 + c_real
        z_imag = 2.0*z_real*z_imag + c_imag;                 // 2ab+c_imag
        z_real = real_temp;
        ++iteration;
    }
    */
       

    //very low, not efficient
    /*
    int iteration = 0;
    int max_iteration = 256;// Iterate up to 255 times, corresponding to 255 colors
    while ((__fmul_rd(z_real, z_real) + __fmul_rd(z_imag, z_imag)) <= 4 && (iteration < max_iteration))
    {
        //Z^2+c
        float real_temp = __fmul_rd(z_real,z_real) - __fmul_rd(z_imag,z_imag) + c_real; // a^2 - b^2 + c_real
        z_imag = __fmul_rd(2.0,__fmul_rd(z_real,z_imag)) + c_imag;                 // 2ab+c_imag
        z_real = real_temp;
        ++iteration;
    }
    */
    
    int factor = (MAX_Iteration - iteration) * 10; //cool
    if (iteration == MAX_Iteration)
    {
        uchar4 output;
        output.w = 0;
        output.x = 0;
        output.y = 0;
        output.z = 255;
        dp_ptr[index] = output;
    }
    else
    {

        uchar4 output;
        output.w = factor;
        output.x = factor;
        output.y = factor;
        output.z = 255;
        dp_ptr[index] = output;
        //dp_ptr[index] = make_uchar4(iteration*0.5, iteration, iteration,255);
    }

    //try to avoid if else, but not so much influence
   /* int factor = (MAX_Iteration-iteration)*10;
    uchar4 output;
    output.w = factor;
    output.x = factor;
    output.y = factor;
    output.z = 255;
    dp_ptr[index] = output;*/

}

void CallingKernel(const float lower_left_real, const float lower_left_imag, float factorX, float factorY, uchar4* dp_ptr)
{
    //use 32*32 thread more slowly

    dim3 blockDim(16, 16, 1);
    dim3 gridDim(WIDTH / blockDim.x, HEIGHT / blockDim.y, 1);
    //printf("grid.x =  %d grid.y =  %d grid.z =  %d\n", gridDim.x, gridDim.y, gridDim.z);
    //printf("block.x =  %d block.y =  %d block.z =  %d\n", blockDim.x, blockDim.y, blockDim.z);
    kernel << < gridDim, blockDim, 0 >> > (lower_left_real, lower_left_imag, factorX, factorY,dp_ptr);
}
